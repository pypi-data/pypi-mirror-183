#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <dirent.h>
#include <iostream>
#include <stdio.h>
#include <string.h>
#include <sys/stat.h>

using namespace std;

#define CUDA_SAFE_CALL(x)                                                         \
{                                                                               \
	if (x != hipSuccess)                                                         \
	{                                                                             \
		printf("  Cuda error %i occured in %s, line %i!\n", x, __FILE__, __LINE__); \
		printf("  Cuda error message:  %s\n", hipGetErrorString(x));               \
		char buffer[256];                                                           \
		cin >> buffer;                                                              \
		exit(-1);                                                                   \
	}                                                                             \
}

typedef unsigned char uchar;

const int size_uchar = sizeof(uchar);

__device__ bool IsNeighborForDistanceTransform(hipSurfaceObject_t track_surf, int _x, int _y, int _z, int depth, int width, int height, int iterations)
{
	bool tag = false;
	uchar track_data;
	surf3Dread(&track_data, track_surf, _x, _y, _z, hipBoundaryModeTrap);
	// track_data = tex3D(text_track_ref, _x, _y, _z);
	if (track_data <= iterations)
	{
		return tag;
	}

	for (int k = _z - 1; k <= _z + 1; k++)
	{
		for (int j = _y - 1; j <= _y + 1; j++)
		{
			for (int i = _x - 1; i <= _x + 1; i++)
			{
				if (((k > 0) && (k < depth)) && ((j > 0) && (j < height)) && ((i > 0) && (i < width)))
				{
					// if (((i+k)-(x+z))*((i+k)-(x+z))!=1) continue;
					// if (((j+k)-(j+z))*((j+k)-(y+z))!=1) continue;
					// int index_neighbor = GetFlat(i, j, k, width, height);
					// if (index_neighbor != index)
					if (i != _x || j != _y || k != _z)
					{
						unsigned char neighbor_value;
						surf3Dread(&neighbor_value, track_surf, i, j, k, hipBoundaryModeTrap);
						// neighbor_value = tex3D(text_track_ref, i, j, k);
						if ((neighbor_value == iterations) && track_data > iterations)
						{
							tag = true;
							return tag;
						}
					}
				}
			}
		}
	}
	return tag;
}

__global__ void DistanceTransform(hipSurfaceObject_t track_surf, int *incluidos, int depth, int width, int height, int iterations)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int z = blockIdx.z * blockDim.z + threadIdx.z;
	if ((x < width) && (y < height) && (z < depth))
	{
		// int i = GetFlat(x, y, z, width, height);
		bool tag = IsNeighborForDistanceTransform(track_surf, x, y, z, depth, width, height, iterations);
		__syncthreads();
		if (tag)
		{
			// track_data[i] = iterations + 1;
			unsigned char tmp_v = iterations + 1;
			if (tmp_v > 254)
			{
				printf("tmp_v:  %d", tmp_v);
			}
			surf3Dwrite(tmp_v, track_surf, x * size_uchar, y, z, hipBoundaryModeTrap);
			*incluidos += 1;
		}
	}
}

__global__ void InitWithLimits(hipSurfaceObject_t track_surf_w, int iteration_limit, int lower_threshold, int upper_threshold, int depth, int width, int height)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int z = blockIdx.z * blockDim.z + threadIdx.z;

	if ((x < width) && (y < height) && (z < depth))
	{
		// int i = GetFlat(x, y, z, width, height);
		unsigned char track_data;
		surf3Dread(&track_data, track_surf_w, x * size_uchar, y, z, hipBoundaryModeTrap);
		// track_data = tex3D(text_track_ref, x, y, z);
		if (track_data >= lower_threshold && track_data < upper_threshold)
		{
	  		// track_data[i] = iteration_limit;
			surf3Dwrite(uchar(iteration_limit), track_surf_w, x * size_uchar, y, z, hipBoundaryModeTrap);
		}
		else
		{
			// track_data[i] = 0;
			surf3Dwrite(uchar(0), track_surf_w, x * size_uchar, y, z, hipBoundaryModeTrap);
		}
	}
}

__global__ void RefactorSeedData(hipSurfaceObject_t tmp_surf, hipSurfaceObject_t seed_surf, int depth, int width, int height)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int z = blockIdx.z * blockDim.z + threadIdx.z;
	if (x >= width || y >= height || z >= depth)
	{
		return;
	}

	// int i = GetFlat(x, y, z, width, height);
	// uchar tmp_data_v = tex3D(text_tmp_ref, x, y, z);
	uchar tmp_data_v;
	surf3Dread(&tmp_data_v, tmp_surf, x * size_uchar, y, z);
	if (tmp_data_v > 0)
	{
		// seed_data[i] = tmp_data_v;
		surf3Dwrite(tmp_data_v, seed_surf, x, y, z);
	}
}

__device__ int IsNeighbor(hipSurfaceObject_t seed_surf, int _x, int _y, int _z, int depth, int width, int height)
{
	uchar max_index = 0;

	// GetCoord(index, &x, &y, &z, width, height);

	for (int k = _z - 1; k <= _z + 1; k++)
	{
		for (int j = _y - 1; j <= _y + 1; j++)
		{
			for (int i = _x - 1; i <= _x + 1; i++)
			{
				if (((k > 0) && (k < depth)) && ((j > 0) && (j < height)) && ((i > 0) && (i < width)))
				{
					// int index_neighbor = GetFlat(i, j, k, width, height);
					// if (index_neighbor != index)
					if (i != _x || j != _y || k != _z)
					{
						// uchar seed_data_v = tex3D(text_seed_ref, i, j, k);
						uchar seed_data_v;
						surf3Dread(&seed_data_v, seed_surf, i, j, k);
						if (seed_data_v > 0)
						{
							return seed_data_v;
						}
					}
				}
			}
		}
	}
	return max_index;
}

__global__ void RegionGrowing(hipSurfaceObject_t tmp_surf, hipSurfaceObject_t seed_surf, hipSurfaceObject_t track_surf_obj, int *incluidos, int depth, int width, int height, int growth_bound)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int z = blockIdx.z * blockDim.z + threadIdx.z;
	if ((x < width) && (y < height) && (z < depth))
	{
		// int i = GetFlat(x, y, z, width, height);
		// uchar seed_data = tex3D(text_seed_ref, x, y, z);
		uchar seed_data;
		surf3Dread(&seed_data, seed_surf, x, y, z);
		if (seed_data != 0)
		{
			return;
		}

		uchar max_index = IsNeighbor(seed_surf, x, y, z, depth, width, height);
		__syncthreads();
		if (max_index > 0)
		{
			uchar tmp_v;
			surf3Dread(&tmp_v, track_surf_obj, x, y, z);
			// if (tex3D(text_track_ref, x, y, z) >= growth_bound)
			if (tmp_v >= growth_bound)
			{
				// tmp_data[i] = max_index;
				surf3Dwrite(max_index, tmp_surf, x * size_uchar, y, z);
				atomicAdd(incluidos, 1);
			}
		}
	}
}

extern "C"
{

	void RegionGrowthEntrance(uchar *h_track_data, uchar *h_seed_data, int width, int height, int depth, int lower_threshold, int upper_threshold, int distance_iteration_limit, int growth_iteration_limit, int gpu)
	{
		hipSetDevice(gpu);
		size_t size_int = sizeof(int);

		// 开辟 GPU 内存: cuda_Array: track
		hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);
		hipArray *d_track_array;
		hipExtent extent3D;
		extent3D.width = width;
		extent3D.depth = depth;
		extent3D.height = height;
		CUDA_SAFE_CALL(hipMalloc3DArray((hipArray **)&d_track_array, &channelDesc, extent3D, hipArraySurfaceLoadStore));

		// 开辟 GPU 内存: cuda_Array: tmp
		hipArray *d_tmp_array;
		CUDA_SAFE_CALL(hipMalloc3DArray((hipArray **)&d_tmp_array, &channelDesc, extent3D, hipArraySurfaceLoadStore));
		// 开辟 GPU 内存: cuda_Array: seed
		hipArray *d_seed_array;
		CUDA_SAFE_CALL(hipMalloc3DArray((hipArray **)&d_seed_array, &channelDesc, extent3D, hipArraySurfaceLoadStore));

		int *h_incluidos = (int *)malloc(size_int);
		int *d_incluidos;
		CUDA_SAFE_CALL(hipMalloc((void **)&d_incluidos, size_int));
		dim3 dimBlock(16, 16, 4);
		dim3 dimGrid(32, 32, (depth + 4) / 4);

		// 将 h_track_data与 d_track_array 绑定
		hipMemcpy3DParms track_parms = {0};
		track_parms.srcPos = make_hipPos(0, 0, 0);
		track_parms.dstPos = make_hipPos(0, 0, 0);
		track_parms.srcPtr = make_hipPitchedPtr(h_track_data, width * size_uchar, width, height);
		track_parms.dstArray = d_track_array;
		track_parms.extent = make_hipExtent(width, height, depth);
		track_parms.kind = hipMemcpyHostToDevice;
		CUDA_SAFE_CALL(hipMemcpy3D(&track_parms));
		// 指定表面内存
		struct hipResourceDesc resDesc;
		memset(&resDesc, 0, sizeof(resDesc));
		resDesc.resType = hipResourceTypeArray;
		// 创建表面内存对象 并绑定 CUDA_Array
		resDesc.res.array.array = d_track_array;
		hipSurfaceObject_t track_surf_obj = 0;
		hipCreateSurfaceObject(&track_surf_obj, &resDesc);

		// 将 h_seed_data与 d_tmp_array 绑定
		hipMemcpy3DParms tmp_parms = {0};
		tmp_parms.srcPos = make_hipPos(0, 0, 0);
		tmp_parms.dstPos = make_hipPos(0, 0, 0);
		tmp_parms.srcPtr = make_hipPitchedPtr(h_seed_data, width * size_uchar, width, height);
		tmp_parms.dstArray = d_tmp_array;
		tmp_parms.extent = make_hipExtent(width, height, depth);
		tmp_parms.kind = hipMemcpyHostToDevice;
		CUDA_SAFE_CALL(hipMemcpy3D(&tmp_parms));
		// 指定表面内存
		struct hipResourceDesc t_resDesc;
		memset(&t_resDesc, 0, sizeof(t_resDesc));
		t_resDesc.resType = hipResourceTypeArray;
		// 创建表面内存对象 并绑定 CUDA_Array
		t_resDesc.res.array.array = d_tmp_array;
		hipSurfaceObject_t tmp_surf_obj = 0;
		hipCreateSurfaceObject(&tmp_surf_obj, &t_resDesc);

		// 将 h_seed_data与 d_seed_array 绑定
		hipMemcpy3DParms seed_parms = {0};
		seed_parms.srcPos = make_hipPos(0, 0, 0);
		seed_parms.dstPos = make_hipPos(0, 0, 0);
		seed_parms.srcPtr = make_hipPitchedPtr(h_seed_data, width * size_uchar, width, height);
		seed_parms.dstArray = d_seed_array;
		seed_parms.extent = make_hipExtent(width, height, depth);
		seed_parms.kind = hipMemcpyHostToDevice;
		CUDA_SAFE_CALL(hipMemcpy3D(&seed_parms));
		// 指定表面内存
		struct hipResourceDesc seed_resDesc;
		memset(&seed_resDesc, 0, sizeof(seed_resDesc));
		seed_resDesc.resType = hipResourceTypeArray;
		// 创建表面内存对象 并绑定 CUDA_Array
		seed_resDesc.res.array.array = d_seed_array;
		hipSurfaceObject_t seed_surf_obj = 0;
		hipCreateSurfaceObject(&seed_surf_obj, &seed_resDesc);

		// DistanceTransform
		InitWithLimits<<<dimGrid, dimBlock>>>(track_surf_obj, distance_iteration_limit, lower_threshold, upper_threshold, depth, width, height);

		int iterations = 0;
		do
		{
			*h_incluidos = 0;
			CUDA_SAFE_CALL(hipMemcpy(d_incluidos, h_incluidos, size_int, hipMemcpyHostToDevice));
			DistanceTransform<<<dimGrid, dimBlock>>>(track_surf_obj, d_incluidos, depth, width, height, iterations);
			CUDA_SAFE_CALL(hipDeviceSynchronize());
			CUDA_SAFE_CALL(hipMemcpy(h_incluidos, d_incluidos, size_int, hipMemcpyDeviceToHost));
			iterations++;
			if (iterations > distance_iteration_limit)
			{
				break;
			}
		} while (*h_incluidos != 0);

		//region_growth
		int limit = 100;
		for (int growth_bound = distance_iteration_limit; growth_bound > 0; growth_bound--)
		{
			iterations = 0;
			if (growth_bound == 1) {
				limit = 0;
			}
			do {
				*h_incluidos = 0;
				CUDA_SAFE_CALL(hipMemcpy(d_incluidos, h_incluidos, size_int, hipMemcpyHostToDevice));
				RegionGrowing<<<dimGrid, dimBlock>>>(tmp_surf_obj, seed_surf_obj, track_surf_obj, d_incluidos, depth, width, height, growth_bound);
				RefactorSeedData<<<dimGrid, dimBlock>>>(tmp_surf_obj, seed_surf_obj, depth, width, height);
				hipDeviceSynchronize();
				CUDA_SAFE_CALL(hipMemcpy(h_incluidos, d_incluidos, size_int, hipMemcpyDeviceToHost));
				if (growth_iteration_limit != -1)
				{
					iterations++;
					if (iterations > growth_iteration_limit)
					{
						break;
					}
				}
			} while (*h_incluidos > limit);
		}

		// CUDA_SAFE_CALL(cudaMemcpy(h_seed_data, d_seed_data, size_space, cudaMemcpyDeviceToHost));

		// cudaExtent extent3d = make_cudaExtent(width, height, depth);
		hipMemcpy3DParms myParms = {0};
		myParms.srcArray = d_seed_array;
		myParms.srcPos = make_hipPos(0, 0, 0);
		myParms.dstPos = make_hipPos(0, 0, 0);
		myParms.dstPtr = make_hipPitchedPtr(h_seed_data, width * size_uchar, width, height);
		myParms.extent = make_hipExtent(width, height, depth);
		myParms.kind = hipMemcpyDeviceToHost;
		CUDA_SAFE_CALL(hipMemcpy3D(&myParms));

		free(h_incluidos);
		// cudaFree(d_seed_data);
		// 解除绑定并释放内存
		hipFreeArray(d_track_array);
		// cudaFree(d_track_data);
		hipDestroySurfaceObject(track_surf_obj);
		hipDestroySurfaceObject(tmp_surf_obj);
		hipDestroySurfaceObject(seed_surf_obj);
		hipFree(d_incluidos);
		// cudaFree(d_tmp_data);
		hipFreeArray(d_tmp_array);
		hipFreeArray(d_seed_array);
	}
}
