#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <dirent.h>
#include <iostream>
#include <stdio.h>
#include <string.h>
#include <sys/stat.h>

using namespace std;

#define CUDA_SAFE_CALL(x)                                                         \
{                                                                               \
	if (x != hipSuccess)                                                         \
	{                                                                             \
		printf("  Cuda error %i occured in %s, line %i!\n", x, __FILE__, __LINE__); \
		printf("  Cuda error message:  %s\n", hipGetErrorString(x));               \
		char buffer[256];                                                           \
		cin >> buffer;                                                              \
		exit(-1);                                                                   \
	}                                                                             \
}

typedef unsigned char uchar;

const int size_uchar = sizeof(uchar);

__device__ bool isBoundary(hipSurfaceObject_t track_surf, int _x, int _y, int _z, int depth, int width, int height, bool interface)
{
	bool tag = false;
	uchar current_value;
	surf3Dread(&current_value, track_surf, _x, _y, _z, hipBoundaryModeTrap);
	if (current_value <= 0)
		return tag;

	for (int k = _z - 1; k <= _z + 1; k++){
		for (int j = _y - 1; j <= _y + 1; j++){
			for (int i = _x - 1; i <= _x + 1; i++){
				if (((k > 0) && (k < depth)) && ((j > 0) && (j < height)) && ((i > 0) && (i < width))){
					if (i != _x || j != _y || k != _z) {
						unsigned char boundary_value;
						surf3Dread(&boundary_value, track_surf, i, j, k, hipBoundaryModeTrap);
						// mask 交接面
						if (interface){
							if((boundary_value != current_value) && (boundary_value != 0)){
								tag = true;
								return tag;
							}
						}
						// mask 表面
						else {
							if (boundary_value != current_value) {
								tag = true;
								return tag;
							}
						}
					} // end if
				} // end if
			} // for i
		} // for j
	} // for k
	return tag;
}


__global__ void SurfaceErodePoint(hipSurfaceObject_t track_surf, int *incluidos, int depth, int width, int height, bool interface)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int z = blockIdx.z * blockDim.z + threadIdx.z;
	if ((x < width) && (y < height) && (z < depth))
	{
		bool boundary_tag = isBoundary(track_surf, x, y, z, depth, width, height, interface);
		__syncthreads();
		if (boundary_tag)
		{
			surf3Dwrite(uchar(0), track_surf, x * size_uchar, y, z, hipBoundaryModeTrap);
			*incluidos += 1;
		}
	}
}


extern "C"
{
void SurfaceErodeEntrance(uchar *h_mask, int width, int height, int depth,  int erode_iterations, bool interface, int gpu)
{
	hipSetDevice(gpu);
	size_t size_int = sizeof(int);

	// 开辟 GPU 内存: cuda_Array: track
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);
	hipArray *d_mask;
	hipExtent extent3D;
	extent3D.width = width;
	extent3D.depth = depth;
	extent3D.height = height;
	CUDA_SAFE_CALL(hipMalloc3DArray((hipArray **)&d_mask, &channelDesc, extent3D, hipArraySurfaceLoadStore));

	int *h_incluidos = (int *)malloc(size_int);
	int *d_incluidos;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_incluidos, size_int));
	dim3 dimBlock(16, 16, 4);
	dim3 dimGrid(32, 32, (depth + 4) / 4);

	// 将 h_mask与 d_mask 绑定
	hipMemcpy3DParms track_parms = {0};
	track_parms.srcPos = make_hipPos(0, 0, 0);
	track_parms.dstPos = make_hipPos(0, 0, 0);
	track_parms.srcPtr = make_hipPitchedPtr(h_mask, width * size_uchar, width, height);
	track_parms.dstArray = d_mask;
	track_parms.extent = make_hipExtent(width, height, depth);
	track_parms.kind = hipMemcpyHostToDevice;
	CUDA_SAFE_CALL(hipMemcpy3D(&track_parms));
	// 指定表面内存
	struct hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	// 创建表面内存对象 并绑定 CUDA_Array
	resDesc.res.array.array = d_mask;
	hipSurfaceObject_t mask_surf_obj = 0;
	hipCreateSurfaceObject(&mask_surf_obj, &resDesc);

	int iterations = 1;
	do
	{
		*h_incluidos = 0;
		CUDA_SAFE_CALL(hipMemcpy(d_incluidos, h_incluidos, size_int, hipMemcpyHostToDevice));
		printf("iteration: %d\n", iterations);
		SurfaceErodePoint<<<dimGrid, dimBlock>>>(mask_surf_obj, d_incluidos, depth, width, height, interface);
		CUDA_SAFE_CALL(hipDeviceSynchronize());
		CUDA_SAFE_CALL(hipMemcpy(h_incluidos, d_incluidos, size_int, hipMemcpyDeviceToHost));
		iterations++;
		if (iterations > erode_iterations)
		{
			break;
		}
	} while (*h_incluidos != 0);

	hipMemcpy3DParms myParms = {0};
	myParms.srcArray = d_mask;
	myParms.srcPos = make_hipPos(0, 0, 0);
	myParms.dstPos = make_hipPos(0, 0, 0);
	myParms.dstPtr = make_hipPitchedPtr(h_mask, width * size_uchar, width, height);
	myParms.extent = make_hipExtent(width, height, depth);
	myParms.kind = hipMemcpyDeviceToHost;
	CUDA_SAFE_CALL(hipMemcpy3D(&myParms));

	free(h_incluidos);
	// 解除绑定并释放内存
	hipFreeArray(d_mask);
	hipDestroySurfaceObject(mask_surf_obj);
	hipFree(d_incluidos);

}   // SurfaceErodeEntrance
}   // extern "C"
